#include "erl_interface.h"
#include "ei.h"    

#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <arpa/inet.h>
#include <getopt.h>

#ifdef GPU_CNODE
extern int dotProduct(float *A, float *B, int len, float *C);
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#else
int dotProduct(float *A, float *B, int len, float *C);
#endif

#include "cnode.h"

int main(int argc, char *argv[]) {

	int port, fd, listen_fd, pub;
	int option = 0;
	ErlConnect conn;
	char *ip_addr, *hostname, *nodename, *fullnodename, *cookie;


	port = 0;
	while ((option = getopt(argc, argv,"p:i:h:n:f:c:")) != -1) {
		switch (option) {
			case 'i' : ip_addr = optarg;
			break;
			case 'h' : hostname = optarg;
	 		break;
     			case 'n' : nodename = optarg; 
	 		break;
			case 'f' : fullnodename = optarg;
			break;
			case 'c' : cookie = optarg;
			break;
			case 'p' : port = atoi(optarg);
			break;
			default: print_usage(); 
			exit(-1);
		}
	}

	if (port == 0) {
		print_usage();
		exit(-1);
	}

	if ((listen_fd = cnode_listen(port)) == -1) {
		erl_err_quit("cnode listen");
	}
	fprintf(stdout, "listening on port %d\n", port);

	erl_init(NULL,0);

	if (cnode_connect(ip_addr, hostname, nodename, fullnodename, cookie) == -1) {
		erl_err_quit("cnode connect");
	}
	fprintf(stdout, "connecting\n");

	if ((pub = erl_publish(port)) == -1) {
		erl_err_quit("erl_publish");
	}
	fprintf(stdout, "publishing on port %d\n", port);

	fprintf(stdout, "waiting on accept...\n");

	while (1) {
		if ((fd = erl_accept(listen_fd, &conn)) == ERL_ERROR) {
			erl_err_quit("erl_accept");
		}

		fprintf(stdout, "Connected to %s\n\r", conn.nodename);

		pid_t parent = getpid();
		pid_t pid = fork();

		if (pid == -1) {
		} else if (pid > 0) {
			close(fd);
			continue;
		} else {

#ifdef GPU_CNODE
    			findCudaDevice(argc, (const char **)argv);
#endif
			cnode_process(fd);
			close(fd);
		}
	}



	exit(0);
}

void print_usage() {
    printf("Usage: cnode -p port -i ip_addr -h hostname -n nodename -f longnodename -c cookie\n");
}

int cnode_connect(char *ip_addr, char *hostname, char *nodename, char *fullnodename, char *cookie) {

	struct in_addr addr;

	//The first argument is the host name.
	//The second argument is the plain node name.
	//The third argument is the full node name.
	//The fourth argument is a pointer to an in_addr struct with the IP address of the host.
	//The fifth argument is the magic cookie.
	//The sixth argument is the instance number.

	addr.s_addr = inet_addr(ip_addr);
	if (erl_connect_xinit(hostname, nodename, fullnodename, &addr, cookie, 0) == -1) {
		erl_err_quit("erl_connect_xinit");
	}

	return 0;
}

int cnode_listen(int port) {
	int listen_fd;
	struct sockaddr_in addr;
	int on = 1;

	if ((listen_fd = socket(AF_INET, SOCK_STREAM, 0)) < 0) {
		return (-1);
	}

	setsockopt(listen_fd, SOL_SOCKET, SO_REUSEADDR, &on, sizeof(on));

	memset((void*) &addr, 0, (size_t) sizeof(addr));
	addr.sin_family = AF_INET;
	addr.sin_port = htons(port);
	addr.sin_addr.s_addr = htonl(INADDR_ANY);

	if (bind(listen_fd, (struct sockaddr*) &addr, sizeof(addr)) < 0)
		return (-1);
	
	listen(listen_fd, 5);

	return listen_fd;
}


void cnode_process(int fd) {

	int loop = 1;                            /* Loop flag */
	int got;                                 /* Result of receive */
	unsigned char buf[BUFSIZE];              /* Buffer for incoming message */
	ErlMessage emsg;                         /* Incoming message */

	ETERM *msg, *fromp, *tuplep, *fnp, *arga, *argb, *resp;
	int res, *static_res_array;
	ETERM *gen_call         = erl_format("~a", "$gen_call");

	while (loop) {

		got = erl_receive_msg(fd, buf, BUFSIZE, &emsg);
		if (got == ERL_TICK) {
			fprintf(stdout, "got an ERL_TICK: ignoring...\n");
		} else if (got == ERL_ERROR) {
			fprintf(stdout, "got an ERL_ERROR: terminating processing...\n");
			loop = 0;
		} else {

			if (emsg.type == ERL_REG_SEND) {
				msg = erl_element(1, emsg.msg);
				fromp = erl_element(2, emsg.msg);
				tuplep = erl_element(3, emsg.msg);

				fnp = erl_element(1, tuplep);
				arga = erl_element(2, tuplep);
				argb = erl_element(3, tuplep);

				if (erl_match(gen_call, msg)) {
					process_gen_call(fd, emsg.from, emsg.msg);
					continue;
				}

				if (strncmp(ERL_ATOM_PTR(fnp), "dot", 3) == 0) {

					int error = 0;
					float *array_a, *array_b;

					if (ERL_IS_CONS(arga) && ERL_IS_CONS(argb) && (erl_length(arga) == erl_length(argb))) {
						array_a = make_float_array(arga);
						array_b = make_float_array(argb);

						int i;

						if (array_a == NULL) {
							error = 1;
						}

						if (array_b == NULL) {
							error = 1;
						}

					} else {
						error = 1;
					}

					if (!error) {
						float C;

						dotProduct(array_a, array_b, erl_length(arga), &C);
						resp = erl_format("{cnode, ~f}", C);
						erl_send(fd, fromp, resp);
					} else {
						ETERM *err;
						err = erl_format("{cnode, ~w}", erl_format("[{err, ~a}]", "error_in_args"));
						erl_send(fd, fromp, err);
						erl_free_compound(err);
					}

					free(array_a);
					free(array_b);

				} else {
					fprintf(stdout, "received unknown atom: %s\n", ERL_ATOM_PTR(fnp));
				}	


				erl_free_term(msg);
				erl_free_term(emsg.from); erl_free_term(emsg.msg);
				erl_free_term(fromp); erl_free_term(tuplep);
				erl_free_term(fnp); erl_free_term(arga); erl_free_term(argb);
				erl_free_term(resp);

				erl_mem_manager_report();
			}
		}
	}
	erl_free_term(gen_call);
	fprintf(stdout,"node terminating\n");
}

static void erl_mem_manager_report() {
	unsigned long allocated, freed;

	erl_eterm_statistics(&allocated,&freed);
	printf("currently allocated blocks: %ld\n",allocated);
	printf("length of freelist: %ld\n",freed);

	/* really free the freelist */
	erl_eterm_release();
}

static void process_gen_call(int fd, ETERM *from, ETERM *msg) {
	ETERM *is_auth  = erl_format("~a", "is_auth");
	ETERM *args = erl_element(3, msg);
	ETERM *arg1 = erl_element(1, args);

	if (erl_match(is_auth, arg1)) {
		ETERM *fromp = erl_element(2, msg);
		ETERM *resp = erl_format("{~w, yes}", erl_element(2, fromp));
		erl_send(fd, from, resp);
		fprintf(stdout, "responded to a ping...\n");
		erl_free_compound(resp);
		erl_free_term(fromp);
	}
	erl_free_term(args);
	erl_free_term(arg1);
	erl_free_term(is_auth);
}


ETERM **array_to_list(int *static_res_array, int length) {

	ETERM **list = (ETERM **)malloc(sizeof(ETERM *) * length);
	int i;

	for(i=0;i<length;i++) {
		list[i] = erl_mk_int(static_res_array[i]);
	}
	return list;
}

void free_list(ETERM **list, int length) {
	int i;

	fprintf(stdout, "freeing: %d\n", length);
	for (i=0;i<length; i++) {
		erl_free_term(list[i]);
	}
	free(list);
}


float *make_float_array(ETERM *argp) {
	int i;
	float *array = (float *)malloc(erl_length(argp)*sizeof(float));
	ETERM *hd, *tl, *nt;

	for (
		i=0, hd=erl_hd(argp), tl=erl_tl(argp);

		i<erl_length(argp);

		i++,

		erl_free_term(hd),
		hd=erl_hd(tl), 
		nt=erl_tl(tl),
		erl_free_term(tl),
		tl=nt)
	{
		if (ERL_IS_INTEGER(hd)) {
			array[i]=(int)ERL_INT_VALUE(hd);
		}
		else if (ERL_IS_FLOAT(hd)) {
			array[i]=ERL_FLOAT_VALUE(hd);
		} else {
			erl_free_term(hd); erl_free_term(tl);
			return (float *)NULL;
		}
	}

	erl_free_term(hd);
	erl_free_term(tl);

	return array;
}

int *make_int_array(ETERM *argp) {
	int i;
	int *array = (int *)malloc(erl_length(argp)*sizeof(int));
	ETERM *hd, *tl, *nt;

	for (
		i=0, hd=erl_hd(argp), tl=erl_tl(argp);

		i<erl_length(argp);

		i++,

		erl_free_term(hd),
		hd=erl_hd(tl), 
		nt=erl_tl(tl),
		erl_free_term(tl),
		tl=nt)
	{
		if (ERL_IS_INTEGER(hd)) {
			array[i]=ERL_INT_VALUE(hd);
		}
		else {
			erl_free_term(hd); erl_free_term(tl);
			return (int *)NULL;
		}
	}

	erl_free_term(hd);
	erl_free_term(tl);

	return array;
}

#ifndef GPU_CNODE
int dotProduct(float *A, float *B, int len, float *C) {
	int i;
	float dp;

	for (i=0;i<len;i++) {
		dp += (A[i]*B[i]);
	}
	*C = dp;
	return 0;
}
#endif
